// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

 //////////////////////////////////////////////////////////////////
 // Implementation of the activation functions for the TCuda      //
 // implementation of the low-level interface.                   //
 //////////////////////////////////////////////////////////////////

#include "TMVA/DNN/Architectures/TCudnn.h"
/*#include "TMVA/DNN/Architectures/Cuda.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"
#include "Kernels.cuh"*/

namespace TMVA
{
namespace DNN
{

//______________________________________________________________________________
template<typename AFloat>
void TCudnn<AFloat>::Activation(TCudaTensor<AFloat> & A, EActivationFunction activFunct, const double coef, const AFloat alpha, const AFloat beta)
{
   hipdnnActivationDescriptor_t activationDescriptor;
   
   CUDNNCHECK(hipdnnCreateActivationDescriptor(&activationDescriptor));
   
   hipdnnActivationMode_t activationMode;
   switch(activFunct) {
      case EActivationFunction::kIdentity: activationMode = HIPDNN_ACTIVATION_PATHTRU;break;
      case EActivationFunction::kRelu:     activationMode = HIPDNN_ACTIVATION_RELU;    break;
      case EActivationFunction::kSigmoid:  activationMode = HIPDNN_ACTIVATION_SIGMOID; break;
      case EActivationFunction::kTanh:     activationMode = HIPDNN_ACTIVATION_TANH;    break;
      // The activations otherwise used are not supported by cuDNN
      default:    activationMode = HIPDNN_ACTIVATION_PATHTRU;     
   };
   CUDNNCHECK(hipdnnSetActivationDescriptor(activationDescriptor,
                                           activationMode,
                                           HIPDNN_PROPAGATE_NAN,
                                           coef));
                                           
   CUDNNCHECK(hipdnnActivationForward(A.GetCudnnHandle(),
                                     activationDescriptor,
                                     &alpha,
                                     A.GetTensorDescriptor(),
                                     A.GetDataPointer(),
                                     &beta,
                                     A.GetTensorDescriptor(),     // Can be computed in place
                                     A.GetDataPointer()));

   CUDNNCHECK(hipdnnDestroyActivationDescriptor(activationDescriptor));
}

//______________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::IdentityDerivative(TCudaTensor<AFloat> & B,
                                           const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::IdentityDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       (int) B.GetNrows(),
       (int) B.GetNcols());
   B.SetComputeStream(s);
}*/

//______________________________________________________________________________
template<typename AFloat>
void TCudnn<AFloat>::Relu(TCudaTensor<AFloat> & A, const double coef, const AFloat alpha, const AFloat beta)
{
   Activation(A, EActivationFunction::kRelu, coef, alpha, beta);
}

//______________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::ReluDerivative(TCudaTensor<AFloat> & B,
                                       const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::ReluDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}*/

//______________________________________________________________________________
template<typename AFloat>
void TCudnn<AFloat>::Sigmoid(TCudaTensor<AFloat> & A, const double coef, const AFloat alpha, const AFloat beta)
{
   Activation(A, EActivationFunction::kSigmoid, coef, alpha, beta);
}

//______________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::SigmoidDerivative(TCudaTensor<AFloat> & B,
                                          const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SigmoidDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}*/

//______________________________________________________________________________
template<typename AFloat>
void TCudnn<AFloat>::Tanh(TCudaTensor<AFloat> & A, const double coef, const AFloat alpha, const AFloat beta)
{
   Activation(A, EActivationFunction::kTanh, coef, alpha, beta);
}

//______________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::TanhDerivative(TCudaTensor<AFloat> & B,
                                       const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::TanhDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}

//______________________________________________________________________________
template<typename AFloat>
void TCudnn<AFloat>::SymmetricRelu(TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SymmetricRelu<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
}

//______________________________________________________________________________
template<typename AFloat>
void TCudnn<AFloat>::SymmetricReluDerivative(TCudaTensor<AFloat> & B,
                                                const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SymmetricReluDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}

//______________________________________________________________________________
template<typename AFloat>
void TCudnn<AFloat>::SoftSign(TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SoftSign<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
}

//______________________________________________________________________________
template<typename AFloat>
void TCudnn<AFloat>::SoftSignDerivative(TCudaTensor<AFloat> & B,
                                           const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SoftSignDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}

//______________________________________________________________________________
template<typename AFloat>
void TCudnn<AFloat>::Gauss(TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::Gauss<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
}

//______________________________________________________________________________
template<typename AFloat>
void TCudnn<AFloat>::GaussDerivative(TCudaTensor<AFloat> & B,
                                    const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::GaussDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}*/

} // namespace DNN
} // namespace TMVA
