// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

 //////////////////////////////////////////////////////////////////
 // Implementation of the activation functions for the TCuda      //
 // implementation of the low-level interface.                   //
 //////////////////////////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cudnn.h"
/*#include "TMVA/DNN/Architectures/Cuda.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"
#include "Kernels.cuh"*/

namespace TMVA
{
namespace DNN
{

//______________________________________________________________________________
template<>
void TCudnn<float>::Activation(TCudaTensor<float> & A, EActivationFunction activFunct, const double coef, const float alpha, const float beta)
{
   hipdnnActivationDescriptor_t activationDescriptor;
   
   CUDNNCHECK(hipdnnCreateActivationDescriptor(&activationDescriptor));
   
   hipdnnActivationMode_t activationMode;
   switch(activFunct) {
      case EActivationFunction::kIdentity: activationMode = HIPDNN_ACTIVATION_PATHTRU;break;
      case EActivationFunction::kRelu:     activationMode = HIPDNN_ACTIVATION_RELU;    break;
      case EActivationFunction::kSigmoid:  activationMode = HIPDNN_ACTIVATION_SIGMOID; break;
      case EActivationFunction::kTanh:     activationMode = HIPDNN_ACTIVATION_TANH;    break;
      // The activations otherwise used are not supported by cuDNN
      default:    activationMode = HIPDNN_ACTIVATION_PATHTRU;     
   };
   CUDNNCHECK(hipdnnSetActivationDescriptor(activationDescriptor,
                                           activationMode,
                                           HIPDNN_PROPAGATE_NAN,
                                           coef));
                                           
   CUDNNCHECK(hipdnnActivationForward(A.GetCudnnHandle(),
                                     activationDescriptor,
                                     &alpha,
                                     A.GetTensorDescriptor(),
                                     A.GetDataPointer(),
                                     &beta,
                                     A.GetTensorDescriptor(),     // Can be computed in place
                                     A.GetDataPointer()));

   CUDNNCHECK(hipdnnDestroyActivationDescriptor(activationDescriptor));
}

template<>
void TCudnn<double>::Activation(TCudaTensor<double> & A, EActivationFunction activFunct, const double coef, const double alpha, const double beta)
{
   hipdnnActivationDescriptor_t activationDescriptor;
   
   CUDNNCHECK(hipdnnCreateActivationDescriptor(&activationDescriptor));
   
   hipdnnActivationMode_t activationMode;
   switch(activFunct) {
      case EActivationFunction::kIdentity: activationMode = HIPDNN_ACTIVATION_PATHTRU;break;
      case EActivationFunction::kRelu:     activationMode = HIPDNN_ACTIVATION_RELU;    break;
      case EActivationFunction::kSigmoid:  activationMode = HIPDNN_ACTIVATION_SIGMOID; break;
      case EActivationFunction::kTanh:     activationMode = HIPDNN_ACTIVATION_TANH;    break;
      // The activations otherwise used are not supported by cuDNN
      default:    activationMode = HIPDNN_ACTIVATION_PATHTRU;     
   };
   CUDNNCHECK(hipdnnSetActivationDescriptor(activationDescriptor,
                                           activationMode,
                                           HIPDNN_PROPAGATE_NAN,
                                           coef));
                                           
   CUDNNCHECK(hipdnnActivationForward(A.GetCudnnHandle(),
                                     activationDescriptor,
                                     &alpha,
                                     A.GetTensorDescriptor(),
                                     A.GetDataPointer(),
                                     &beta,
                                     A.GetTensorDescriptor(),     // Can be computed in place
                                     A.GetDataPointer()));

   CUDNNCHECK(hipdnnDestroyActivationDescriptor(activationDescriptor));
}

//______________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::IdentityDerivative(TCudaTensor<AFloat> & B,
                                           const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::IdentityDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       (int) B.GetNrows(),
       (int) B.GetNcols());
   B.SetComputeStream(s);
}*/

//______________________________________________________________________________
template<>
void TCudnn<float>::Relu(TCudaTensor<float> & A, const double coef, const float alpha, const float beta)
{
   Activation(A, EActivationFunction::kRelu, coef, alpha, beta);
}

template<>
void TCudnn<double>::Relu(TCudaTensor<double> & A, const double coef, const double alpha, const double beta)
{
   Activation(A, EActivationFunction::kRelu, coef, alpha, beta);
}

//______________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::ReluDerivative(TCudaTensor<AFloat> & B,
                                       const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::ReluDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}*/

//______________________________________________________________________________
template<>
void TCudnn<float>::Sigmoid(TCudaTensor<float> & A, const double coef, const float alpha, const float beta)
{
   Activation(A, EActivationFunction::kSigmoid, coef, alpha, beta);
}

template<>
void TCudnn<double>::Sigmoid(TCudaTensor<double> & A, const double coef, const double alpha, const double beta)
{
   Activation(A, EActivationFunction::kSigmoid, coef, alpha, beta);
}

//______________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::SigmoidDerivative(TCudaTensor<AFloat> & B,
                                          const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SigmoidDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}*/

//______________________________________________________________________________
template<>
void TCudnn<float>::Tanh(TCudaTensor<float> & A, const double coef, const float alpha, const float beta)
{
   Activation(A, EActivationFunction::kTanh, coef, alpha, beta);
}

template<>
void TCudnn<double>::Tanh(TCudaTensor<double> & A, const double coef, const double alpha, const double beta)
{
   Activation(A, EActivationFunction::kTanh, coef, alpha, beta);
}

//______________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::TanhDerivative(TCudaTensor<AFloat> & B,
                                       const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::TanhDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}*/

//______________________________________________________________________________
template<>
void TCudnn<float>::SymmetricRelu(TCudaTensor<float> & A, const double coef, const float alpha, const float beta)
{
   Activation(A, EActivationFunction::kIdentity, coef, alpha, beta);
}

template<>
void TCudnn<double>::SymmetricRelu(TCudaTensor<double> & A, const double coef, const double alpha, const double beta)
{
   Activation(A, EActivationFunction::kIdentity, coef, alpha, beta);
}
//______________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::SymmetricReluDerivative(TCudaTensor<AFloat> & B,
                                                const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SymmetricReluDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}*/

//______________________________________________________________________________
template<>
void TCudnn<float>::SoftSign(TCudaTensor<float> & A, const double coef, const float alpha, const float beta)
{
   Activation(A, EActivationFunction::kIdentity, coef, alpha, beta);
}

template<>
void TCudnn<double>::SoftSign(TCudaTensor<double> & A, const double coef, const double alpha, const double beta)
{
   Activation(A, EActivationFunction::kIdentity, coef, alpha, beta);
}

//______________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::SoftSignDerivative(TCudaTensor<AFloat> & B,
                                           const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SoftSignDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}*/

//______________________________________________________________________________
template<>
void TCudnn<float>::Gauss(TCudaTensor<float> & A, const double coef, const float alpha, const float beta)
{
   Activation(A, EActivationFunction::kIdentity, coef, alpha, beta);
}

template<>
void TCudnn<double>::Gauss(TCudaTensor<double> & A, const double coef, const double alpha, const double beta)
{
   Activation(A, EActivationFunction::kIdentity, coef, alpha, beta);
}

//______________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::GaussDerivative(TCudaTensor<AFloat> & B,
                                    const TCudaTensor<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::GaussDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}*/

} // namespace DNN
} // namespace TMVA
