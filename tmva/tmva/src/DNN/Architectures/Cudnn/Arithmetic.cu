// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

///////////////////////////////////////////////////////////////////
// Contains additional arithmetic functions required by the CUDA //
// neural network implementation.                                //
///////////////////////////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cudnn.h"
#include "TMVA/DNN/Architectures/Cuda/CudaTensor.h"
//#include "TMVA/DNN/Architectures/Cuda/CudaMatrix.h"
//#include "TMVA/DNN/Architectures/Cuda/Device.h"
//#include "../Cuda/Kernels.cuh"

namespace TMVA
{
namespace DNN
{

//____________________________________________________________________________
template<>
void TCudnn<float>::Multiply(TCudaTensor<float> &C,
                             const TCudaTensor<float> &A,
                             const TCudaTensor<float> &B,
                             const float alpha,
                             const float beta,
                             const float gamma)
{   
                  
   // Descriptor for the Tensor Operation
   hipdnnOpTensorDescriptor_t opTensorDescr;
   CUDNNCHECK(hipdnnCreateOpTensorDescriptor(&opTensorDescr));
   
   CUDNNCHECK(hipdnnSetOpTensorDescriptor(opTensorDescr,
                                         HIPDNN_OP_TENSOR_MUL,
                                         HIPDNN_DATA_FLOAT,
                                         HIPDNN_PROPAGATE_NAN)); // NaN will be propagated
  
   // C = MUL(alpha*A, beta*B) + gamma*C                                          
   hipdnnStatus_t status = hipdnnOpTensor(A.GetCudnnHandle(),
                            opTensorDescr,
                            &alpha,
                            A.GetTensorDescriptor(),
                            A.GetDataPointer(),
                            &beta,
                            B.GetTensorDescriptor(),
                            B.GetDataPointer(),
                            &gamma,           // gamma = 0: Don't add C
                            C.GetTensorDescriptor(),
                            C.GetDataPointer());
                                                    
   CUDNNCHECK(hipdnnDestroyOpTensorDescriptor(opTensorDescr));
}

//____________________________________________________________________________
template<>
void TCudnn<double>::Multiply(TCudaTensor<double> &C,
                             const TCudaTensor<double> &A,
                             const TCudaTensor<double> &B,                            
                             const double alpha,
                             const double beta,
                             const double gamma)
{                         
   // Descriptor for the Tensor Operation
   hipdnnOpTensorDescriptor_t opTensorDescr;
   CUDNNCHECK(hipdnnCreateOpTensorDescriptor(&opTensorDescr));
   
   CUDNNCHECK(hipdnnSetOpTensorDescriptor(opTensorDescr,
                                         HIPDNN_OP_TENSOR_MUL,
                                         HIPDNN_DATA_DOUBLE,
                                         HIPDNN_PROPAGATE_NAN)); // NaN will be propagated

   // C = MUL(alpha*A, beta*B) + gamma*C                                          
   CUDNNCHECK(hipdnnOpTensor(A.GetCudnnHandle(),
                            opTensorDescr,
                            &alpha,
                            A.GetTensorDescriptor(),
                            A.GetDataPointer(),
                            &beta,
                            B.GetTensorDescriptor(),
                            B.GetDataPointer(),
                            &gamma,           // gamma = 0: Don't add C
                            C.GetTensorDescriptor(),
                            C.GetDataPointer()));
                                                    
   CUDNNCHECK(hipdnnDestroyOpTensorDescriptor(opTensorDescr));
}

//____________________________________________________________________________
/*template<>
void TCudnn<float>::TransposeMultiply(TCudaTensor<float> & C,
                                      const TCudaTensor<float> & A,
                                      const TCudaTensor<float> & B)
{

}*/
//____________________________________________________________________________
/*template<>
void TCudnn<double>::TransposeMultiply(TCudaTensor<double> & C,
                                      const TCudaTensor<double> & A,
                                      const TCudaTensor<double> & B)
{

}*/

//____________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::Hadamard(TCudaTensor<AFloat> & B,
                             const TCudaTensor<AFloat> &A)
{

}*/

//____________________________________________________________________________
template<>
float TCudnn<float>::Sum(const TCudaTensor<float> & A, const float alpha, const float beta)
{
   hipdnnHandle_t cudnnHandle = A.GetCudnnHandle();

   // The output tensor C, which has dimensions of a number
   TCudaHostBuffer<float>    hostBuffer (1);
   const std::vector<size_t> shapeVec {1,1,1,1};
   // This constructor copies the data automatically to device
   TCudaTensor<float>        C (1, hostBuffer, 4, shapeVec);
                                         
   // Descriptor for the Tensor Reduction
   hipdnnReduceTensorDescriptor_t reduceTensorDescr;
   CUDNNCHECK(hipdnnCreateReduceTensorDescriptor(&reduceTensorDescr));
   CUDNNCHECK(hipdnnSetReduceTensorDescriptor(reduceTensorDescr,
                                             HIPDNN_REDUCE_TENSOR_ADD,
                                             HIPDNN_DATA_FLOAT,
                                             HIPDNN_PROPAGATE_NAN,                // NaN will be propagated
                                             HIPDNN_REDUCE_TENSOR_FLATTENED_INDICES,
                                             //HIPDNN_REDUCE_TENSOR_NO_INDICES,     // Do not compute indices
                                             HIPDNN_32BIT_INDICES));              // Type of the indices
                                             
   // Find the minimum size of the indices
   size_t indiceSizeInBytes;
   void*  indices = nullptr;
   CUDNNCHECK(cudnnGetReductionIndicesSize(cudnnHandle,
                                           reduceTensorDescr,
                                           A.GetTensorDescriptor(),
                                           C.GetTensorDescriptor(),
                                           &indiceSizeInBytes));
   hipMalloc(&indices, indiceSizeInBytes);
   
   // Find the minimum size of the workspace needed for the reduction
   size_t workspaceSizeInBytes;
   void*  workspace = nullptr;
   CUDNNCHECK(hipdnnGetReductionWorkspaceSize(cudnnHandle,
                                             reduceTensorDescr,
                                             A.GetTensorDescriptor(),
                                             C.GetTensorDescriptor(),
                                             &workspaceSizeInBytes));
   hipMalloc(&workspace, workspaceSizeInBytes);
                                         
   // Tensor reduction to the dimensions of the tensor C set above
   // C = alpha * reduce op ( A ) + beta * C                                 
   CUDNNCHECK(hipdnnReduceTensor(cudnnHandle,
                                reduceTensorDescr,
                                indices,
                                indiceSizeInBytes,
                                workspace,
                                workspaceSizeInBytes,
                                &alpha,
                                A.GetTensorDescriptor(),
                                A.GetDataPointer(),
                                &beta,
                                C.GetTensorDescriptor(),
                                C.GetDataPointer()));
                                
   // Get return value from device
   TCudaDeviceBuffer<float>& resultDeviceBuffer = C.GetDeviceBuffer();
   resultDeviceBuffer.CopyTo(hostBuffer);
               
   hipFree(indices);          
   hipFree(workspace);
   CUDNNCHECK(hipdnnDestroyReduceTensorDescriptor(reduceTensorDescr));
   
   return *hostBuffer;
}

//____________________________________________________________________________
template<>
double TCudnn<double>::Sum(const TCudaTensor<double> & A, const double alpha, const double beta)
{
   hipdnnHandle_t cudnnHandle = A.GetCudnnHandle();

   // The output tensor C, which has dimensions of a number
   TCudaHostBuffer<double>   hostBuffer (1);
   const std::vector<size_t> shapeVec {1,1,1,1};
   // This constructor copies the data automatically to device
   TCudaTensor<double>       C (1, hostBuffer, 4, shapeVec);
                                         
   // Descriptor for the Tensor Reduction
   hipdnnReduceTensorDescriptor_t reduceTensorDescr;
   CUDNNCHECK(hipdnnCreateReduceTensorDescriptor(&reduceTensorDescr));
   CUDNNCHECK(hipdnnSetReduceTensorDescriptor(reduceTensorDescr,
                                             HIPDNN_REDUCE_TENSOR_ADD,
                                             HIPDNN_DATA_DOUBLE,
                                             HIPDNN_PROPAGATE_NAN,                // NaN will be propagated
                                             HIPDNN_REDUCE_TENSOR_NO_INDICES,     // Do not compute indices
                                             HIPDNN_32BIT_INDICES));              // Type of the indices
                                             
   // Find the minimum size of the indices
   size_t indiceSizeInBytes;
   void*  indices{nullptr};
   CUDNNCHECK(cudnnGetReductionIndicesSize(cudnnHandle,
                                           reduceTensorDescr,
                                           A.GetTensorDescriptor(),
                                           C.GetTensorDescriptor(),
                                           &indiceSizeInBytes));
   hipMalloc(&indices, indiceSizeInBytes);
   
   // Find the minimum size of the workspace needed for the reduction
   size_t workspaceSizeInBytes;
   void*  workspace{nullptr};
   CUDNNCHECK(hipdnnGetReductionWorkspaceSize(cudnnHandle,
                                             reduceTensorDescr,
                                             A.GetTensorDescriptor(),
                                             C.GetTensorDescriptor(),
                                             &workspaceSizeInBytes));
   hipMalloc(&workspace, workspaceSizeInBytes);
                                         
   // Tensor reduction to the dimensions of the tensor C set above
   // C = alpha * reduce op ( A ) + beta * C                                 
   CUDNNCHECK(hipdnnReduceTensor(cudnnHandle,
                                reduceTensorDescr,
                                indices,
                                indiceSizeInBytes,
                                workspace,
                                workspaceSizeInBytes,
                                &alpha,
                                A.GetTensorDescriptor(),
                                A.GetDataPointer(),
                                &beta,
                                C.GetTensorDescriptor(),
                                C.GetDataPointer()));
                                
   // Get return value from device
   TCudaDeviceBuffer<double>& resultDeviceBuffer = C.GetDeviceBuffer();
   resultDeviceBuffer.CopyTo(hostBuffer);
   
   hipFree(indices);          
   hipFree(workspace);
   CUDNNCHECK(hipdnnDestroyReduceTensorDescriptor(reduceTensorDescr));
   
   return *hostBuffer;
}


//____________________________________________________________________________
template<>
float TCudnn<float>::Sum(const std::vector<TCudaTensor<float> > &A,
                              const float alpha,
                              const float beta)
{
   float totalSum = 0.0;
   for (size_t i = 0; i < A.size(); ++i) {
      totalSum += Sum(A[i], alpha, beta);
   }
   return totalSum;
}

//____________________________________________________________________________
template<>
double TCudnn<double>::Sum(const std::vector<TCudaTensor<double> > &A,
                              const double alpha,
                              const double beta)
{
   double totalSum = 0.0;
   for (size_t i = 0; i < A.size(); ++i) {
      totalSum += Sum(A[i], alpha, beta);
   }
   return totalSum;
}

//____________________________________________________________________________
/*template<>
void TCudnn<float>::SumColumns(TCudaTensor<float> & B,
                              const TCudaTensor<float> & A)
{

}*/

//____________________________________________________________________________
/*template<>
void TCudnn<double>::SumColumns(TCudaTensor<double> & B,
                               const TCudaTensor<double> & A)
{

}

template<>
void TCudnn<float>::SumRows(TCudaTensor<float> & B,
                           const TCudaTensor<float> & A)
{

}*/

//____________________________________________________________________________
/*template<>
void TCudnn<double>::SumRows(TCudaTensor<double> & B,
                             const TCudaTensor<double> & A)
{

}*/

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// \brief Checks two matrices for element-wise equality.
/// \tparam AFloat An architecture-specific floating point number type.
/// \param A The first matrix.
/// \param B The second matrix.
/// \param epsilon Equality tolerance, needed to address floating point arithmetic.
/// \return Whether the two matrices can be considered equal element-wise
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template<typename AFloat>
bool TCudnn<AFloat>::AlmostEquals(const TCudaTensor<AFloat> &A, const TCudaTensor<AFloat> &B, double epsilon)
{

}

//____________________________________________________________________________
template<>
void TCudnn<float>::ScaleAdd(TCudaTensor<float> & B,
                             const TCudaTensor<float> & A,
                             const float alpha,
                             const float beta)
{
   CUDNNCHECK(hipdnnAddTensor(A.GetCudnnHandle(),
                             &alpha,
                             A.GetTensorDescriptor(),
                             A.GetDataPointer(),
                             &beta,
                             B.GetTensorDescriptor(),        // Destination Tensor
                             B.GetDataPointer()));
}

//____________________________________________________________________________
template<>
void TCudnn<double>::ScaleAdd(TCudaTensor<double> & B,
                              const TCudaTensor<double> & A,
                              const double alpha,
                              const double beta)
{
   CUDNNCHECK(hipdnnAddTensor(A.GetCudnnHandle(),
                             &alpha,
                             A.GetTensorDescriptor(),
                             A.GetDataPointer(),
                             &beta,
                             B.GetTensorDescriptor(),        // Destination Tensor
                             B.GetDataPointer()));
}

//____________________________________________________________________________
template<>
void TCudnn<float>::ScaleAdd(std::vector<TCudaTensor<float>> & B,
                             const std::vector<TCudaTensor<float>> & A,
                             const float alpha,
                             const float beta)
{
   for (size_t i = 0; i < A.size(); ++i) {
      ScaleAdd(B[i], A[i], alpha, beta);
   }
}

//____________________________________________________________________________
template<>
void TCudnn<double>::ScaleAdd(std::vector<TCudaTensor<double>> & B,
                              const std::vector<TCudaTensor<double>> & A,
                              const double alpha,
                              const double beta)
{
   for (size_t i = 0; i < A.size(); ++i) {
      ScaleAdd(B[i], A[i], alpha, beta);
   }
}

//____________________________________________________________________________
template<>
void TCudnn<double>::ConstAdd(TCudaTensor<double> &A, const double beta)
{
   // tmp tensor that does the addition
   TCudaTensor<double> C (A);
   C.SetConstVal(beta);
   
   ScaleAdd(A, C);
}

//____________________________________________________________________________
template<>
void TCudnn<float>::ConstAdd(TCudaTensor<float> &A, const float beta)
{
   // tmp tensor that does the addition
   TCudaTensor<float> C (A);
   C.SetConstVal(beta);
   
   ScaleAdd(A, C);
}

//____________________________________________________________________________
template<>
void TCudnn<double>::ConstMult(TCudaTensor<double> &A, const double beta)
{   
   CUDNNCHECK(hipdnnScaleTensor(A.GetCudnnHandle(),
                               A.GetTensorDescriptor(),
                               A.GetDataPointer(),
                               &beta));
}

//____________________________________________________________________________
template<>
void TCudnn<float>::ConstMult(TCudaTensor<float> &A, const float beta)
{   
   CUDNNCHECK(hipdnnScaleTensor(A.GetCudnnHandle(),
                               A.GetTensorDescriptor(),
                               A.GetDataPointer(),
                               &beta));
}

//____________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::ReciprocalElementWise(TCudaTensor<AFloat> &A)
{

}*/

//____________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::SquareElementWise(TCudaTensor<AFloat> &A)
{

}*/

//____________________________________________________________________________
template<>
void TCudnn<float>::SqrtElementWise(TCudaTensor<float> &A, const float alpha, const float beta, const float gamma)
{
   // Descriptor for the Tensor Operation
   hipdnnOpTensorDescriptor_t opTensorDescr;
   CUDNNCHECK(hipdnnCreateOpTensorDescriptor(&opTensorDescr));
   
   CUDNNCHECK(hipdnnSetOpTensorDescriptor(opTensorDescr,
                                         HIPDNN_OP_TENSOR_SQRT,
                                         HIPDNN_DATA_FLOAT,
                                         HIPDNN_PROPAGATE_NAN)); // NaN will be propagated
                                         
   // C = MUL(alpha*A, beta*B) + gamma*C                                    
   CUDNNCHECK(hipdnnOpTensor(A.GetCudnnHandle(),
                            opTensorDescr,
                            &alpha,
                            A.GetTensorDescriptor(),
                            A.GetDataPointer(),
                            &beta,
                            A.GetTensorDescriptor(),
                            A.GetDataPointer(),
                            &gamma,
                            A.GetTensorDescriptor(),  // Save result in A
                            A.GetDataPointer()));
                            
   CUDNNCHECK(hipdnnDestroyOpTensorDescriptor(opTensorDescr));
}

//____________________________________________________________________________
template<>
void TCudnn<double>::SqrtElementWise(TCudaTensor<double> &A, const double alpha, const double beta, const double gamma)
{   
   // Descriptor for the Tensor Operation
   hipdnnOpTensorDescriptor_t opTensorDescr;
   CUDNNCHECK(hipdnnCreateOpTensorDescriptor(&opTensorDescr));
   
   CUDNNCHECK(hipdnnSetOpTensorDescriptor(opTensorDescr,
                                         HIPDNN_OP_TENSOR_SQRT,
                                         HIPDNN_DATA_DOUBLE,
                                         HIPDNN_PROPAGATE_NAN)); // NaN will be propagated
                                         
   // C = MUL(alpha*A, beta*B) + gamma*C                                    
   CUDNNCHECK(hipdnnOpTensor(A.GetCudnnHandle(),
                            opTensorDescr,
                            &alpha,
                            A.GetTensorDescriptor(),
                            A.GetDataPointer(),
                            &beta,
                            A.GetTensorDescriptor(),
                            A.GetDataPointer(),
                            &gamma,
                            A.GetTensorDescriptor(),  // Save result in A
                            A.GetDataPointer()));
                            
   CUDNNCHECK(hipdnnDestroyOpTensorDescriptor(opTensorDescr));
}

/// Adam updates 
//____________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::AdamUpdate(TCudaTensor<AFloat> &A, const TCudaTensor<AFloat> & M, const TCudaTensor<AFloat> & V, AFloat alpha, AFloat eps)
{

}*/

//____________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::AdamUpdateFirstMom(TCudaTensor<AFloat> &A, const TCudaTensor<AFloat> & B, AFloat beta)
{

}*/

//____________________________________________________________________________
/*template<typename AFloat>
void TCudnn<AFloat>::AdamUpdateSecondMom(TCudaTensor<AFloat> &A, const TCudaTensor<AFloat> & B, AFloat beta)
{

}*/
   
} // DNN
} // TMVA
