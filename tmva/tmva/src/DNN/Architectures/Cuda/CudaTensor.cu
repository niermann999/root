// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

/////////////////////////////////////////////
// Implementation of the TCudaTensor class. //
/////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cuda/CudaTensor.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"

#include <cassert>

namespace TMVA {
namespace DNN  {


// Static members.
//____________________________________________________________________________
template<typename AFloat>
size_t                   TCudaTensor<AFloat>::fInstances        = 0;
/*template<typename AFloat>
hipblasHandle_t           TCudaTensor<AFloat>::fCublasHandle     = nullptr;*/
template<typename AFloat>
hipdnnHandle_t            TCudaTensor<AFloat>::fCudnnHandle      = nullptr;
template<typename AFloat>
hipdnnTensorDescriptor_t  TCudaTensor<AFloat>::fTensorDescriptor = nullptr;
template<typename AFloat>
hipdnnDataType_t          TCudaTensor<AFloat>::fDataType         = HIPDNN_DATA_FLOAT;
/*template<typename AFloat>
AFloat                   * TCudaTensor<AFloat>::fDeviceReturn   = nullptr;*/
/*template<typename AFloat>
AFloat                   * TCudaTensor<AFloat>::fOnes           = nullptr;*/
/*template<typename AFloat>
hiprandState_t            * TCudaTensor<AFloat>::fCurandStates   = nullptr;*/
/*template<typename AFloat>
size_t                   TCudaTensor<AFloat>::fNCurandStates    = 0;*/
/*template<typename AFloat>
size_t                   TCudaTensor<AFloat>::fNOnes            = 0;*/

// Constructors.
//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor()
    : fShape(), fStrides(nullptr), fNDim(0), fSize(0), fElementBuffer()
{
   InitializeCuda();
}

//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(size_t size, size_t dim, const std::vector<size_t> shape)
    : fNDim(dim), fSize(size), fElementBuffer(size, 0)
{
   // Need a shape array with at least 4 entries for cuDNN tensors
   if (fNDim < 2) {
       std::puts("No matching cuDNN tensor description for given input dimension(s). "
                 "Inputs should be given as: batch size, no. channels, image dimensions.");
       exit(EXIT_FAILURE);
   }
   // fNDim contains only the spacial tensor dimensions, batchsize and #channels are
   // contained in the shape array
   size_t shape_size = fShape.size();
   // Reduce shape size afterwards for loop and direct array access
   fStrides = new size_t[shape_size--];
   for (int i = 0; i < shape_size; ++i) {
       fStrides[i] = shape[i+1];
       for (int j = 0; j < i; j++) {
          fStrides[j] *= shape[i+1];
       }
   }
   // Last stride should be one for cudnn
   fStrides[shape_size] = 1;
   
   assert(fSize == fStrides[0]*shape[0]);
   
   InitializeCuda();
}

//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(size_t size, const AFloat * host_data, size_t dim, const std::vector<size_t> shape)
    : TCudaTensor(size, dim, shape)
{
   // do I need to allocate this buffer ???? 
   // is not a mem leak
   // AFloat * buffer = new AFloat[fSize];
   // size_t index = 0;
   // for (size_t j = 0; j < fSize; ++j) {
   //       buffer[j] = static_cast<AFloat>(host_data[j]);
   //    }
   // }

   hipMemcpy(fElementBuffer, host_data, fSize * sizeof(AFloat),
              hipMemcpyHostToDevice);
}

//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(TCudaDeviceBuffer<AFloat> buffer, size_t dim, const std::vector<size_t> shape)
    : fNDim(dim), fElementBuffer(buffer), fShape(std::move(shape))
{   
   // Need a shape array with at least 4 entries for cuDNN tensors
   if (fNDim < 2) {
       std::puts("No matching cuDNN tensor description for given input dimension(s). "
                 "Inputs should be given as: batch size, no. channels, image dimensions.");
       exit(EXIT_FAILURE);
   }
   // fNDim contains only the spacial tensor dimensions, batchsize and #channels are
   // contained in the shape array
   size_t shape_size = fShape.size();
   // Reduce shape size afterwards for loop and direct array access
   fStrides = new size_t[shape_size--];
   for (int i = 0; i < shape_size; ++i) {
       fStrides[i] = shape[i+1];
       for (int j = 0; j < i; j++) {
          fStrides[j] *= shape[i+1];
       }
   }
   // Last stride should be one for cudnn
   fStrides[shape_size] = 1;
   
   fSize = fStrides[0]*shape[0];
   
   InitializeCuda();  
}

//____________________________________________________________________________
template <typename AFloat>
TCudaTensor<AFloat>::~TCudaTensor() {

   // Free resources of this instance
   if (fStrides) delete[] fStrides;
      
   if (--fInstances <= 0) {
      hipdnnDestroyTensorDescriptor(fTensorDescriptor);
      hipdnnDestroy(fCudnnHandle);
   }
}

//____________________________________________________________________________
template <typename AFloat>
inline void TCudaTensor<AFloat>::InitializeCuda()
{
   // add further initialization than done in TMatrixcPU::iNITIALIZEcUDA
   if (fInstances == 0) {
      //hipblasCreate(&fCublasHandle);
      CUDNNCHECK(hipdnnCreate(&fCudnnHandle));
   //     CUDACHECK(hipMalloc(& fDeviceReturn, sizeof(AFloat)));
   //     CUDACHECK(hipMalloc(& fCurandStates, TDevice::NThreads(*this)));
   
   
      CUDNNCHECK(hipdnnCreateTensorDescriptor(&fTensorDescriptor));
   }
   // if (TDevice::NThreads(*this) > (int) fNCurandStates) {
   //     fNCurandStates = TDevice::NThreads(*this);
   //     if (fCurandStates) {
   //         hipFree(fCurandStates);
   //     }
   //     hipMalloc(&fCurandStates, TDevice::NThreads(*this) * sizeof(hiprandState_t));
   //     InitializeCurandStates();
   // }
   
   fInstances++;
      
   if      (std::is_same<AFloat, double>::value) { fDataType = HIPDNN_DATA_DOUBLE; }
   else if (std::is_same<AFloat, float>::value)  { fDataType = HIPDNN_DATA_FLOAT; }
   
   if (!fStrides) {
      return;
   }
   // cuDNN NdTensor format has a minsize of 3 tensor dimensions
   else if (fNDim == 2) {
      CUDNNCHECK(hipdnnSetTensor4dDescriptor(fTensorDescriptor,
                                            HIPDNN_TENSOR_NCHW,// Layout of the tensor in memory
                                            fDataType,
                                            (int)fShape[0],  // batch size
                                            (int)fShape[1],  // no. channels
                                            (int)fShape[2],  // image height
                                            (int)fShape[3]));// image width
   
   }
   // Evade case fNDim = 0
   else {
     CUDNNCHECK(hipdnnSetTensorNdDescriptor(fTensorDescriptor,
                                            fDataType,
                                            (int)fNDim,
                                            (int *)fShape.data(),
                                            (int *)fStrides));
   }
}

//____________________________________________________________________________
template<typename AFloat>
void TCudaTensor<AFloat>::InitializeCurandStates()
{
   // dim3 blockDims = TDevice::BlockDims2D();
   // dim3 gridDims  = TDevice::GridDims2D(*this);
   // CurandInitializationKernel<<<gridDims, blockDims>>>(time(nullptr), fCurandStates);
}

#if 0
// Conversion to RTensor
//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::operator Experimental::RTensor<AFloat>() const
{
   std::vector<size_t> shape(fNDims, fNDims + fDim)
   
   Experimental::RTensor<AFloat> hostTensor( shape)

   AFloat * buffer = new AFloat[fSize];
   hipMemcpy(buffer, fElementBuffer, fSize * sizeof(AFloat),
              hipMemcpyDeviceToHost);

   int index = 0;
   for (int j = 0; j < fSize; j++) {
         hostTensor.GetData()[j] = static_cast<AFloat>(buffer[j]);
      }
   }

   delete[] buffer;
   return hostTensor;
}
#endif
// Explicit Instantiations.

template class TCudaTensor<float>;
template class TCudaTensor<double>;

} // namespace DNN
} // namespace TMVA
